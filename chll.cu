#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "murmur3.cu"

// Lengths of an input line, a 36 character UUID
#define INPUT_LINE_WIDTH 36

// Lengths of an process line, a 36 character
// UUID followed by 4 padding bytes for alignment
#define PROCESS_LINE_WIDTH 40

// Default enough space for 100K entries
#define INITIAL_SIZE 100000

// Multiple the buffer size by 4 each time
#define BUF_MULT 4

// Width of our hash function output
#define HASH_WIDTH 16

// Number of threads per block
#define THREADS_PER_BLOCK 192

// Define the number of HLL buckets, and prefix bits to use
#define HLL_BUCKETS 1024
#define HLL_PREFIX_BITS 10  // log2(HLL_BUCKETS)

// How wide is each bucket
#define HLL_BUCKET_WIDTH 6
#define HLL_MAX_SCAN 64     // 2**HLL_BUCKET_WIDTH

#define TWO_32 4294967296 // 2**32

static double alpha() {
    return 0.7213/(1 + 1.079/HLL_BUCKETS);
}


static double raw_estimate(unsigned int *inp) {
    double multi = alpha() * HLL_BUCKETS * HLL_BUCKETS;
    double inv_sum = 0;
    for (int i=0; i<HLL_BUCKETS;i++) {
        inv_sum += 1 / pow(2.0, (int)inp[i]);
    }
    return (1 / inv_sum) * multi;
}


static double range_corrected(double raw, unsigned int *inp) {
    if (raw < (5/2)*HLL_BUCKETS) {
        int numzero = 0;
        for (int i=0; i < HLL_BUCKETS; i++) {
            if (inp[i] == 0) numzero++;
        }
        if (numzero == 0)
            return raw;
        else
            return HLL_BUCKETS * log(HLL_BUCKETS / numzero);

    } else if (raw > (1/30)*TWO_32) {
        return -1*TWO_32*log(1 - (raw / TWO_32));
    } else {
        return raw;
    }
}


__host__ int read_input(char **inp, int *inp_len) {
    // Get the initial buffer
    int buf_size = INITIAL_SIZE * PROCESS_LINE_WIDTH;
    char *buf = (char*)malloc(buf_size);

    int offset = 0;
    int in;
    while (1) {
        in = read(STDIN_FILENO, buf+offset, INPUT_LINE_WIDTH+1);
        if (in == 0) break;
        else if (in < 0) {
            perror("Failed to read input!\n");
            free(buf);
            return 1;
        } else if (in == INPUT_LINE_WIDTH + 1) {
            offset += INPUT_LINE_WIDTH;
            *(buf+offset) = 0;
            *(buf+offset+1) = 0;
            *(buf+offset+2) = 0;
            *(buf+offset+3) = 0;
            offset += 4;
        } else {
            printf("Input is not %d byte aligned!\n", INPUT_LINE_WIDTH);
            free(buf);
            return 1;
        }

        // Check if we need to resize
        if (offset + PROCESS_LINE_WIDTH >= buf_size) {
            char *new_buf = (char*)malloc(buf_size * BUF_MULT);
            memcpy(new_buf, buf, offset);
            free(buf);
            buf = new_buf;
            buf_size *= BUF_MULT;
        }
    }

    // Return points for data
    *inp_len = offset;
    *inp = buf;
    return 0;
}


// Performs a single pass over the data to hash it
__global__ void hash_data(int n, char *in, char *out) {
    int offset = (blockIdx.x * blockDim.x + threadIdx.x);
    if (offset < n) {
        MurmurHash3_x64_128(in + (offset * PROCESS_LINE_WIDTH), INPUT_LINE_WIDTH, 0, out + (offset * HASH_WIDTH));
    }
}


// Performs a single pass over the data to extract
// the bucket and position of each element
__global__ void extract_hll(int n, char *in, char *out) {
    int offset = (blockIdx.x * blockDim.x + threadIdx.x);
    if (offset < n) {
        uint64_t *hash = (uint64_t*)(in + (HASH_WIDTH * offset));

        // Get the first HLL_PREFIX_BITS to determine the bucket
        int bucket = hash[0] >> (64 - HLL_PREFIX_BITS);

        // Finds the position of the least significant 1 (0 to 64)
        int position = __ffsll(hash[1]);

        // Adjust for the limit of the bucket
        if (position == 0) {
            position = HLL_MAX_SCAN - 1;
        } else
            position = min(position, HLL_MAX_SCAN) - 1;

        // Update the output
        uint16_t *outp = ((uint16_t*)out) + offset;
        *outp = ((bucket << HLL_BUCKET_WIDTH) | position);
    }
}


// Uses a two dimensional grid to build the HLL
__global__ void build_hll(int n, uint16_t *in, unsigned int *out) {
    __shared__ int maxPos;
    maxPos = 0;
    __syncthreads();
    int offset = (blockIdx.y * blockDim.y + blockIdx.x * blockDim.x + threadIdx.x);

    if (offset < n) {
        // Extract the parts
        uint16_t val = *(in + offset);
        int bucket = val >> HLL_BUCKET_WIDTH;

        // Only continue if the bucket matches the y index
        if (bucket != blockIdx.y) return;

        // Update the maximum position
        int pos = val & ((1 << HLL_BUCKET_WIDTH) - 1);
        atomicMax(&maxPos, pos);

        // Wait for all the maximums to be sync'd
        __syncthreads();
        atomicMax(&out[blockIdx.y], maxPos);
    }
}


__host__ int main(int argc, char **argv) {
    // Read the input
    printf("Reading input...\n");
    char *inp;
    int inp_len;
    if (read_input(&inp, &inp_len) || inp_len == 0)
        return 1;

    // Move the data to the GPU
    printf("Copying to GPU...\n");
    char *gpu_in, *hashed;
    hipMalloc((void**)&gpu_in, inp_len);
    hipMemcpy(gpu_in, inp, inp_len, hipMemcpyHostToDevice);

    // Determine block sets
    int n = inp_len / PROCESS_LINE_WIDTH;
    hipMalloc((void**)&hashed, HASH_WIDTH * n);
    int blocks = ceil((float)n / (float)THREADS_PER_BLOCK);

    // Hash all the data for the HLL construction
    printf("Hashing data... (%d lines, %d blocks, %d threads)\n", n, blocks, THREADS_PER_BLOCK);
    hash_data<<<blocks, THREADS_PER_BLOCK>>>(n, gpu_in, hashed);
    hipError_t res = hipDeviceSynchronize();
    if (res != hipSuccess) {
        printf("Hashing failed: %s\n", hipGetErrorString(res));
        return 1;
    }

    // Extract the HLL's values
    printf("Extracting HLL values...\n");
    char *hll_vals;
    hipMalloc((void**)&hll_vals, n * 2);
    extract_hll<<<blocks, THREADS_PER_BLOCK>>>(n, hashed, hll_vals);
    res = hipDeviceSynchronize();
    if (res != hipSuccess) {
        printf("HLL extraction failed: %s\n", hipGetErrorString(res));
        return 1;
    }

    // Build the HLL's
    printf("Building HLL...\n");
    int hll_size = HLL_BUCKETS * sizeof(unsigned int);
    unsigned int *hll;
    unsigned int *host_hll = (unsigned int*)malloc(hll_size);
    memset(host_hll, 0, hll_size);
    hipMalloc((void**)&hll, hll_size);
    hipMemcpy(hll, host_hll, hll_size, hipMemcpyHostToDevice);

    dim3 dimGrid(blocks, HLL_BUCKETS);
    build_hll<<<dimGrid, THREADS_PER_BLOCK>>>(n, (uint16_t*)hll_vals, hll);
    res = hipDeviceSynchronize();
    if (res != hipSuccess) {
        printf("HLL construction failed: %s\n", hipGetErrorString(res));
        return 1;
    }

    // Copy the HLL back
    hipMemcpy(host_hll, hll, hll_size, hipMemcpyDeviceToHost);

    // Estimate cardinality
    printf("Estimating cardinality...\n");
    double raw = raw_estimate(host_hll);
    double adj = range_corrected(raw, host_hll);
    printf("Est: %0.1f Raw: %0.1f\n", adj, raw);

    // Cleanup
    printf("Cleanup...\n");
    free(inp);
    hipFree(gpu_in);
    hipFree(hashed);

    return 0;
}

